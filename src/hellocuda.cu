

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main() {
    hipError_t cudaerr;
    float* deviceA;
    int result = 0;

    size_t available, total;
    hipMemGetInfo(&available, &total);


    if(result = (hipMalloc((void **) &deviceA, 10 * sizeof(float)) != hipSuccess))
	printf("Failed to malloc deviceA, errorCode:%d\n", result);

    cuda_hello<<<1,1>>>();

    cudaerr = hipDeviceSynchronize();

    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error %d \"%s\".\n",
	       cudaerr,
               hipGetErrorString(cudaerr));
    return 0;
}


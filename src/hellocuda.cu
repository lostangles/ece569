

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main() {
    hipError_t cudaerr;
    
    cuda_hello<<<1,1>>>();

    cudaerr = hipDeviceSynchronize();

    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error %d \"%s\".\n",
	       cudaerr,
               hipGetErrorString(cudaerr));
    return 0;
}

